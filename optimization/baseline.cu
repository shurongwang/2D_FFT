#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define loop(i, to) for (int i = 0; i < to; ++i)

const int T = 16;

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S) {
	/*
	Modify this function to implement the forward pass described in Chapter 16.
	We have added an additional dimension to the tensors to support an entire mini-batch
	The goal here is to be correct AND fast.

	Function paramter definitions:
	output - output
	input - input
	mask - convolution kernel
	B - batch_size (number of images in x)
	M - number of output feature maps
	C - number of input feature maps
	H - input height dimension
	W - input width dimension
	K - kernel height and width (K x K)
	S - stride step length
	*/

	const int H_out = (H - K) / S + 1;
	const int W_out = (W - K) / S + 1;
	
	// (void)H_out; // silence declared but never referenced warning. remove this line when you start working
	// (void)W_out; // silence declared but never referenced warning. remove this line when you start working

	// We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
	// An example use of these macros:
	// float a = in_4d(0,0,0,0)
	// out_4d(0,0,0,0) = a

	#define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
	#define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
	#define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

	// Insert your GPU convolution kernel code here
	
	// int b = blockIdx.y, m = threadIdx.y;
	// int h = blockIdx.x, w = threadIdx.x;

	int b = blockIdx.x / M, m = blockIdx.x % M;
	int h = blockIdx.y * T + threadIdx.x;
	int w = blockIdx.z * T + threadIdx.y;

	float res = 0;
	loop (c, C) loop (p, K) loop (q, K) {
		res += in_4d(b, c, h * S + p, w * S + q) * mask_4d(m, c, p, q);
	}
	
	if (h < H_out && w < W_out) out_4d(b, m, h, w) = res;

	#undef out_4d
	#undef in_4d
	#undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S) {
	// Allocate memory and copy over the relevant data structures to the GPU

	// We pass double pointers for you to initialize the relevant device pointers,
	//  which are passed to the other two functions.

	// Useful snippet for error checking
	// hipError_t error = hipGetLastError();
	// if(error != hipSuccess)
	// {
	//     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
	//     exit(-1);
	// }
   
	const int H_out = (H - K) / S + 1;
	const int W_out = (W - K) / S + 1;
	const int in_size = B * C * H * W;
	const int out_size = B * M * H_out * W_out;
	const int mask_size = M * C * K * K;

	hipMalloc((void**) &(*device_output_ptr), out_size * sizeof(float));
	hipMalloc((void**) &(*device_input_ptr), in_size * sizeof(float));
	hipMalloc((void**) &(*device_mask_ptr), mask_size * sizeof(float));

	hipMemcpy(*device_input_ptr, host_input, in_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S) {
	// Set the kernel dimensions and call the kernel

	const int H_out = (H - K) / S + 1;
	const int W_out = (W - K) / S + 1;
	
	dim3 Grid(B * M, ceil(float(H_out) / T), ceil(float(W_out) / T)); 
	dim3 Block(T, T, 1);
	
	conv_forward_kernel<<<Grid, Block>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S) {
	// Copy the output back to host
	const int H_out = (H - K) / S + 1;
	const int W_out = (W - K) / S + 1;
	const int out_size = B * M * H_out * W_out;

	hipMemcpy(host_output, device_output, out_size * sizeof(float), hipMemcpyDeviceToHost);
   
	// Free device memory
	hipFree(device_output);
	hipFree(device_input);
	hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	for(int dev = 0; dev < deviceCount; dev++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
		std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
		std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
		std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
		std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
		std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
		std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
		std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
		std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
	}
}
